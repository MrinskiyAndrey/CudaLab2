#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "wb.h"

//@@ Сложение векторов
__global__ void vecAdd(float* in1, float* in2, float* out, int len) 
{  
    int idx = threadIdx.x;             //Получаем id текущей нити.
    out[idx] = in1[idx] + in2[idx];    //Расчитываем результат.
}

int main(int argc, char** argv) {
    wbArg_t args;
    int inputLength;
    float* hostInput1;
    float* hostInput2;
    float* hostOutput;
    float* deviceInput1;
    float* deviceInput2;
    float* deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float*)malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    //@@ Выделение памяти GPU
    wbTime_start(GPU, "Allocating GPU memory.");
    hipMalloc(&deviceInput1, inputLength * sizeof(float));
    hipMalloc(&deviceInput2, inputLength * sizeof(float));
    hipMalloc(&deviceOutput, inputLength * sizeof(float));
    wbTime_stop(GPU, "Allocating GPU memory.");

    //@@ Копирование памяти на GPU
    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(GPU, "Copying input memory to the GPU.");

    //@@ Инициализируйте размерности сетки и блоков
    int blockSize = 128;
    int gridSize = ceil(static_cast<double>(inputLength) / blockSize);

    //@@ Запустите ядро GPU
    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd <<<gridSize, blockSize>>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    //@@ Скопируйте память GPU обратно на хост
    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying output memory to the CPU");

    //@@ Освободите память GPU
    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);
    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}
